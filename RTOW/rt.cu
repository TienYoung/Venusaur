#include "hip/hip_runtime.h"
﻿#include <optix.h>

#include <OptiXToolkit/ShaderUtil/color.h>

#include "rt.h"

extern "C" {
	__constant__ Params params;
}

extern "C" __global__ void __raygen__uv()
{
	const uint3 launch_index = optixGetLaunchIndex();
	int i = launch_index.x;
	int j = launch_index.y;

	int image_width = params.width;
	int image_height = params.height;

	auto r = double(i) / (image_width - 1);
	auto g = double(j) / (image_height - 1);
	auto b = 0.0;

	params.image[j * image_width + i] = make_color(make_float3(r, g, b));
}